#include "hip/hip_runtime.h"
﻿/**
 * Nearest neighbor search
 * マップ内に、店、工場などのゾーンがある確率で配備されている時、
 * 住宅ゾーンから直近の店、工場までのマンハッタン距離を計算する。
 *
 * 各店、工場から周辺に再帰的に距離を更新していくので、O(N)で済む。
 * しかも、GPUで並列化することで、さらに計算時間を短縮できる。
 *
 * shared memoryを使用して高速化できるか？
 */

#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <list>
#include <time.h>
#include "ZonePlanMCMC2.cuh"

__host__ __device__
unsigned int rand(unsigned int* randx) {
    *randx = *randx * 1103515245 + 12345;
    return (*randx)&2147483647;
}

__host__ __device__
float randf(unsigned int* randx) {
	return rand(randx) / (float(2147483647) + 1);
}

__host__ __device__
float randf(unsigned int* randx, float a, float b) {
	return randf(randx) * (b - a) + a;
}

__host__ __device__
int sampleFromCdf(unsigned int* randx, float* cdf, int num) {
	float rnd = randf(randx, 0, cdf[num-1]);

	for (int i = 0; i < num; ++i) {
		if (rnd <= cdf[i]) return i;
	}

	return num - 1;
}

__host__ __device__
int sampleFromPdf(unsigned int* randx, float* pdf, int num) {
	if (num == 0) return 0;

	float cdf[40];
	cdf[0] = pdf[0];
	for (int i = 1; i < num; ++i) {
		if (pdf[i] >= 0) {
			cdf[i] = cdf[i - 1] + pdf[i];
		} else {
			cdf[i] = cdf[i - 1];
		}
	}

	return sampleFromCdf(randx, cdf, num);
}

/**
 * ゾーンプランを生成する。
 */
__host__
void generateZoningPlan(zone_plan& zoningPlan, std::vector<float> zoneTypeDistribution) {
	std::vector<float> numRemainings(NUM_FEATURES + 1);
	for (int i = 0; i < NUM_FEATURES + 1; ++i) {
		numRemainings[i] = CITY_SIZE * CITY_SIZE * zoneTypeDistribution[i];
	}

	unsigned int randx = 0;

	for (int r = 0; r < CITY_SIZE; ++r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			int type = sampleFromPdf(&randx, numRemainings.data(), numRemainings.size());
			zoningPlan.zones[r][c].type = type;
			numRemainings[type] -= 1;
		}
	}
}

/**
 * 直近の店までの距離を計算する（マルチスレッド版、shared memory使用）
 */
__global__
void computeDistanceToStore(zone_plan* zoningPlan, DistanceMap* distanceMap) {
	__shared__ int sDist[(int)(GPU_BLOCK_SIZE * GPU_BLOCK_SCALE)][(int)(GPU_BLOCK_SIZE * GPU_BLOCK_SCALE)][NUM_FEATURES];
	__shared__ uint3 sQueue[QUEUE_MAX + 1];
	__shared__ unsigned int queue_begin;
	__shared__ unsigned int queue_end;

	queue_begin = 0;
	queue_end = 0;
	__syncthreads();

	// global memoryからshared memoryへコピー
	int num_strides = (GPU_BLOCK_SIZE * GPU_BLOCK_SIZE * GPU_BLOCK_SCALE * GPU_BLOCK_SCALE + GPU_NUM_THREADS - 1) / GPU_NUM_THREADS;
	int r0 = blockIdx.y * GPU_BLOCK_SIZE - GPU_BLOCK_SIZE * (GPU_BLOCK_SCALE - 1) * 0.5;
	int c0 = blockIdx.x * GPU_BLOCK_SIZE - GPU_BLOCK_SIZE * (GPU_BLOCK_SCALE - 1) * 0.5;
	for (int i = 0; i < num_strides; ++i) {
		int r1 = (i * GPU_NUM_THREADS + threadIdx.x) / (int)(GPU_BLOCK_SIZE * GPU_BLOCK_SCALE);
		int c1 = (i * GPU_NUM_THREADS + threadIdx.x) % (int)(GPU_BLOCK_SIZE * GPU_BLOCK_SCALE);

		// これ、忘れてた！！
		if (r1 >= GPU_BLOCK_SIZE * GPU_BLOCK_SCALE || c1 >= GPU_BLOCK_SIZE * GPU_BLOCK_SCALE) continue;

		int type = 0;
		if (r0 + r1 >= 0 && r0 + r1 < CITY_SIZE && c0 + c1 >= 0 && c0 + c1 < CITY_SIZE) {
			type = zoningPlan->zones[r0 + r1][c0 + c1].type;
		}
		for (int feature_id = 0; feature_id < NUM_FEATURES; ++feature_id) {
			distanceMap->distances[r0 + r1][c0 + c1][feature_id] = MAX_DIST;
			if (type - 1 == feature_id) {
				sDist[r1][c1][feature_id] = 0;
				unsigned int q_index = atomicInc(&queue_end, QUEUE_MAX);
				sQueue[q_index] = make_uint3(c1, r1, feature_id);
			} else {
				sDist[r1][c1][feature_id] = MAX_DIST;
			}
		}
	}
	__syncthreads();

	// 距離マップを生成
	unsigned int q_index;
	while ((q_index = atomicInc(&queue_begin, QUEUE_MAX)) < queue_end) {
	//while (queue_begin < queue_end) {
		uint3 pt = sQueue[q_index];
		if (pt.x < 0 || pt.x >= GPU_BLOCK_SIZE * GPU_BLOCK_SCALE || pt.y < 0 || pt.y >= GPU_BLOCK_SIZE * GPU_BLOCK_SCALE) continue;

		int d = sDist[pt.y][pt.x][pt.z];

		if (pt.y > 0) {
			unsigned int old = atomicMin(&sDist[pt.y-1][pt.x][pt.z], d + 1);
			if (old > d + 1) {
				q_index = atomicInc(&queue_end, QUEUE_MAX);
				sQueue[q_index] = make_uint3(pt.x, pt.y-1, pt.z);
			}
		}
		if (pt.y < CITY_SIZE - 1) {
			unsigned int old = atomicMin(&sDist[pt.y+1][pt.x][pt.z], d + 1);
			if (old > d + 1) {
				q_index = atomicInc(&queue_end, QUEUE_MAX);
				sQueue[q_index] = make_uint3(pt.x, pt.y+1, pt.z);
			}
		}
		if (pt.x > 0) {
			unsigned int old = atomicMin(&sDist[pt.y][pt.x-1][pt.z], d + 1);
			if (old > d + 1) {
				q_index = atomicInc(&queue_end, QUEUE_MAX);
				sQueue[q_index] = make_uint3(pt.x-1, pt.y, pt.z);
			}
		}
		if (pt.x < CITY_SIZE - 1) {
			unsigned int old = atomicMin(&sDist[pt.y][pt.x+1][pt.z], d + 1);
			if (old > d + 1) {
				q_index = atomicInc(&queue_end, QUEUE_MAX);
				sQueue[q_index] = make_uint3(pt.x+1, pt.y, pt.z);
			}
		}
	}

	__syncthreads();

	// global memoryの距離マップへ、コピーする
	for (int i = 0; i < num_strides; ++i) {
		int r1 = (i * GPU_NUM_THREADS + threadIdx.x) / (int)(GPU_BLOCK_SIZE * GPU_BLOCK_SCALE);
		int c1 = (i * GPU_NUM_THREADS + threadIdx.x) % (int)(GPU_BLOCK_SIZE * GPU_BLOCK_SCALE);

		// これ、忘れてた！！
		if (r1 >= GPU_BLOCK_SIZE * GPU_BLOCK_SCALE || c1 >= GPU_BLOCK_SIZE * GPU_BLOCK_SCALE) continue;

		if (r0 + r1 >= 0 && r0 + r1 < CITY_SIZE && c0 + c1 >= 0 && c0 + c1 < CITY_SIZE) {
			for (int feature_id = 0; feature_id < NUM_FEATURES; ++feature_id) {
				atomicMin(&distanceMap->distances[r0 + r1][c0 + c1][feature_id], sDist[r1][c1][feature_id]);
			}
		}
	}
}

/**
 * 直近の店までの距離を計算する（CPU版）
 */
__host__
void computeDistanceToStoreCPU(zone_plan* zoningPLan, DistanceMap* distanceMap) {
	std::list<int3> queue;

	for (int feature_id = 0; feature_id < NUM_FEATURES; ++feature_id) {
		for (int cell_id = 0; cell_id < CITY_SIZE * CITY_SIZE; ++cell_id) {
			int r = cell_id / CITY_SIZE;
			int c = cell_id % CITY_SIZE;

			if (zoningPLan->zones[r][c].type - 1 == feature_id) {
				queue.push_back(make_int3(c, r, feature_id));
				distanceMap->distances[r][c][feature_id] = 0;
			} else {
				distanceMap->distances[r][c][feature_id] = MAX_DIST;
			}
		}
	}

	while (!queue.empty()) {
		int3 pt = queue.front();
		queue.pop_front();

		int d = distanceMap->distances[pt.y][pt.x][pt.z];

		if (pt.y > 0) {
			if (distanceMap->distances[pt.y-1][pt.x][pt.z] > d + 1) {
				distanceMap->distances[pt.y-1][pt.x][pt.z] = d + 1;
				queue.push_back(make_int3(pt.x, pt.y-1, pt.z));
			}
		}
		if (pt.y < CITY_SIZE - 1) {
			if (distanceMap->distances[pt.y+1][pt.x][pt.z] > d + 1) {
				distanceMap->distances[pt.y+1][pt.x][pt.z] = d + 1;
				queue.push_back(make_int3(pt.x, pt.y+1, pt.z));
			}
		}
		if (pt.x > 0) {
			if (distanceMap->distances[pt.y][pt.x-1][pt.z] > d + 1) {
				distanceMap->distances[pt.y][pt.x-1][pt.z] = d + 1;
				queue.push_back(make_int3(pt.x-1, pt.y, pt.z));
			}
		}
		if (pt.x < CITY_SIZE - 1) {
			if (distanceMap->distances[pt.y][pt.x+1][pt.z] > d + 1) {
				distanceMap->distances[pt.y][pt.x+1][pt.z] = d + 1;
				queue.push_back(make_int3(pt.x+1, pt.y, pt.z));
			}
		}
	}
}

/**
 * デバッグ用に、ゾーンプランを表示する。
 */
__host__
void showDevZoningPlan(zone_plan* zoningPlan) {
	zone_plan plan;

	CUDA_CALL(hipMemcpy(&plan, zoningPlan, sizeof(zone_plan), hipMemcpyDeviceToHost));
	for (int r = CITY_SIZE - 1; r >= 0; --r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			printf("%d, ", plan.zones[r][c].type);
		}
		printf("\n");
	}
	printf("\n");
}

/**
 * デバッグ用に、ゾーンプランを表示する。
 */
__host__
void showDevDistMap(DistanceMap* distMap, int feature_id) {
	DistanceMap map;

	CUDA_CALL(hipMemcpy(&map, distMap, sizeof(DistanceMap), hipMemcpyDeviceToHost));
	for (int r = CITY_SIZE - 1; r >= 0; --r) {
		for (int c = 0; c < CITY_SIZE; ++c) {
			printf("%d, ", map.distances[r][c][feature_id]);
		}
		printf("\n");
	}
	printf("\n");
}

__host__
void zonePlanMCMCGPUfunc2(zone_plan** hostZoningPlan, int numIterations)
{
	time_t start, end;


	*hostZoningPlan = (zone_plan*)malloc(sizeof(zone_plan));
	DistanceMap* hostDistanceMap = (DistanceMap*)malloc(sizeof(DistanceMap));
	DistanceMap* hostDistanceMap2 = (DistanceMap*)malloc(sizeof(DistanceMap));

	// 距離を初期化
	//memset(hostDistanceMap, MAX_DIST, sizeof(DistanceMap));
	//memset(hostDistanceMap2, MAX_DIST, sizeof(DistanceMap));

	std::vector<float> zoneTypeDistribution(6);
	zoneTypeDistribution[0] = 0.5f;
	zoneTypeDistribution[1] = 0.2f;
	zoneTypeDistribution[2] = 0.1f;
	zoneTypeDistribution[3] = 0.1f;
	zoneTypeDistribution[4] = 0.05f;
	zoneTypeDistribution[5] = 0.05f;
	
	// 初期プランを生成
	start = clock();
	generateZoningPlan(**hostZoningPlan, zoneTypeDistribution);
	end = clock();
	printf("generateZoningPlan: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);
	
	// 初期プランをデバイスバッファへコピー
	zone_plan* devZoningPlan;
	CUDA_CALL(hipMalloc((void**)&devZoningPlan, sizeof(zone_plan)));
	CUDA_CALL(hipMemcpy(devZoningPlan, *hostZoningPlan, sizeof(zone_plan), hipMemcpyHostToDevice));

	// デバッグ用に、初期プランを表示
	if (CITY_SIZE <= 100) {
		showDevZoningPlan(devZoningPlan);
	}

	// 距離マップ用に、デバイスバッファを確保
	DistanceMap* devDistanceMap;
	CUDA_CALL(hipMalloc((void**)&devDistanceMap, sizeof(DistanceMap)));

	///////////////////////////////////////////////////////////////////////
	// CPU版で、直近の店までの距離を計算
	/*
	start = clock();
	for (int iter = 0; iter < 1000; ++iter) {
		computeDistanceToStoreCPU(hostZoningPlan, hostDistanceMap2);
	}
	end = clock();
	printf("computeDistanceToStore CPU: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);
	*/

	int* devQueueStart;
	CUDA_CALL(hipMalloc((void**)&devQueueStart, sizeof(int)));
	int* devQueueEnd;
	CUDA_CALL(hipMalloc((void**)&devQueueEnd, sizeof(int)));

	printf("start...\n");

	///////////////////////////////////////////////////////////////////////
	// warmp up
	computeDistanceToStore<<<dim3(CITY_SIZE / GPU_BLOCK_SIZE, CITY_SIZE / GPU_BLOCK_SIZE), GPU_NUM_THREADS>>>(devZoningPlan, devDistanceMap);

	// マルチスレッドで、直近の店までの距離を計算
	start = clock();
	for (int iter = 0; iter < 1000; ++iter) {
		computeDistanceToStore<<<dim3(CITY_SIZE / GPU_BLOCK_SIZE, CITY_SIZE / GPU_BLOCK_SIZE), GPU_NUM_THREADS>>>(devZoningPlan, devDistanceMap);
		hipDeviceSynchronize();
	}
	end = clock();
	printf("computeDistanceToStore GPU: %lf\n", (double)(end-start)/CLOCKS_PER_SEC);



	CUDA_CALL(hipMemcpy(devZoningPlan, *hostZoningPlan, sizeof(zone_plan), hipMemcpyHostToDevice));
	showDevZoningPlan(devZoningPlan);
	computeDistanceToStore<<<dim3(CITY_SIZE / GPU_BLOCK_SIZE, CITY_SIZE / GPU_BLOCK_SIZE), GPU_NUM_THREADS>>>(devZoningPlan, devDistanceMap);
	hipDeviceSynchronize();



	// 距離マップを表示
	showDevDistMap(devDistanceMap, 1);



	// release device buffer
	hipFree(devZoningPlan);
	hipFree(devDistanceMap);

	// release host buffer
	//free(hostZoningPlan);
	free(hostDistanceMap);
	free(hostDistanceMap2);

	//hipDeviceReset();
}
